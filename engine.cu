#include "hip/hip_runtime.h"
#include <chrono>
#include "engine.h"
#include "bvh_node.h"

__device__ Color castRayAndCalcColor(WorldRecord* worldRecord, const Ray& ray, const u32 maxDepth)
{
	BvhNode* bvhRootNodePtr = worldRecord->getBvhRootNodeDevicePtr();
	Color resultColor(0xFFFFFF);
	Ray currentRay = ray;
	
	for (u32 depth = 0; depth < maxDepth; depth++)
	{
		HitRecord record;
		if (bvhRootNodePtr->isHit(currentRay, 0.001f, MAXFLOAT, record))
		{
			// レコードには本当に衝突したオブジェクトの情報が一部入っているので、
			// その情報を基にレコードを正確に更新する。
			{
				//衝突座標の設定
				record.position = currentRay.pointAt(record.t);

				//法線の設定
				const Vec4 normal(record.normal, 0.0f);
				const Mat4& invTransposeTransformMat = record.hitObject->getTransform().getInvTransposeTransformMatrix();
				record.normal = (invTransposeTransformMat * normal).extractXYZ().normalize();
			}
            
			Ray scatteredRay;
			const Color emissionFromObject = record.material->emission(0, 0, record.position) * record.hitObject->getSurfaceProperty().getAlbedo();
			Color albedo(0x000000);
			f32 pdf = 0.0f;
			if (record.material->scatter(currentRay, record, albedo, scatteredRay, pdf))
			{
				resultColor = emissionFromObject + resultColor* albedo * record.material->scatteringPdf(ray, record, scatteredRay);
				currentRay = scatteredRay;
			}
			else
			{
				return emissionFromObject * resultColor;
			}
		}
		else
		{
			return Color(0x000000);
		}
	}

	return Color(0x000000);
}

__global__ void castRayToWorld(WorldRecord* worldRecord, Color* pixels, Camera* camera, const u32 screenSizeW, const u32 screenSizeH, const u32 sampleSize, const u32 maxDepth)
{
	const u32 id_w = blockIdx.x * blockDim.x + threadIdx.x;
	const u32 id_h = blockIdx.y * blockDim.y + threadIdx.y;
	if (id_h % 100 == 0 && id_w % 100 == 0) printf("%d , %d\n", id_w, id_h);
	if (id_w >= screenSizeW || id_h >= screenSizeH)
	{
		return;
	}

	const u32 pixelIndex = id_h * screenSizeW + id_w;
	
		const f32 inv_screenSizeW = 1.0f / static_cast<f32>(screenSizeW - 1);
		const f32 inv_screenSizeH = 1.0f / static_cast<f32>(screenSizeH - 1);
	
		Color resultColor = Color(0x000000);
		for (u32 s = 0; s < sampleSize; s++)
		{
			const f32 samplingRange = 0.01f;
		const f32 u = static_cast<f32>(id_w + RandomGeneratorGPU::signed_uniform_real() * samplingRange) * inv_screenSizeW;
		const f32 v = static_cast<f32>(id_h + RandomGeneratorGPU::signed_uniform_real() * samplingRange) * inv_screenSizeH;
			
			Ray ray = camera->getRay(u, v);
	
			resultColor += castRayAndCalcColor(worldRecord, ray, maxDepth);
		}
		resultColor /= sampleSize;

		
		*(pixels + pixelIndex) = resultColor;
}

void RayTracingEngine::render(World& world, RenderTarget& renderTarget, const u32 sampleSize, const u32 depth)
{
	std::chrono::system_clock::time_point start, end;
	start = std::chrono::system_clock::now();
    printf("Rendering Start\n");



	dim3 block(16, 16);
	dim3 grid((renderTarget.getResolutionWidth() + block.x - 1) / block.x, (renderTarget.getResolutionHeight() + block.y - 1) / block.y);
	castRayToWorld << <grid, block >> > (
		world.getWorldRecordDevicePtr(),
		renderTarget.getPixels(), 
		world.getCameraManagedPtr(), 
		renderTarget.getResolutionWidth(), 
		renderTarget.getResolutionHeight(),
		sampleSize,
		depth);

    KERNEL_ERROR_CHECKER;

    printf("Rendering Finish\n");
	end = std::chrono::system_clock::now();
	f32 time = static_cast<f32>(std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / 1000.0);
	printf("Rendering Time = %fs\n", time);
}