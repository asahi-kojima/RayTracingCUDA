#include "hip/hip_runtime.h"
#include <algorithm>
#include "engine.h"


RayTracingEngine::RayTracingEngine()
{
	hipMalloc(&mCamera, sizeof(Camera));
}

RayTracingEngine::~RayTracingEngine()
{
	hipDeviceSynchronize();
	hipFree(mCamera);
}










__global__ void getCenters(Hittable** world, size_t objectNum, vec3* centerList)
{
	for (u32 i = 0; i < objectNum; i++)
	{
		centerList[i] = world[i]->calcAABB().getCenterPos();
	}
}


void sort_along_axis(std::vector<std::pair<vec3, u32>>& pairs, const u32 start, u32 end, u32 depth = 0)
{
	if (end - start == 1)
	{
		return;
	}

	std::sort(pairs.begin() + start, pairs.begin() + end, [depth](std::pair<vec3, u32>& pair0, std::pair<vec3, u32> pair1) {const u32 axis_of_sort = depth % 3; return pair0.first[axis_of_sort] < pair1.first[axis_of_sort]; });

	sort_along_axis(pairs, start, start + (end - start) / 2, depth + 1);
	sort_along_axis(pairs, start + (end - start) / 2, end, depth + 1);
}


void sortObjects(Hittable** world, size_t objectNum, u32* indexList)
{
	printf("Sort of World object start\n");
	
	//collect center info of objects;
	vec3 *centerList;
	CHECK(hipMallocManaged(&centerList, sizeof(vec3) * objectNum));

	getCenters<<<1,1>>>(world, objectNum, centerList);
	GPU_ERROR_CHECKER(hipPeekAtLastError());
	CHECK(hipDeviceSynchronize());

	//sort
	std::vector<std::pair<vec3, u32> > pairs;
	for (u32 i = 0; i < objectNum; i++)
	{
		pairs.push_back({centerList[i], i});
	}

	//std::sort(pairs.begin(), pairs.end());
	sort_along_axis(pairs,0, objectNum);

	for (u32 i = 0; i < objectNum; i++)
	{
		indexList[i] = pairs[i].second;
	}


	hipFree(centerList);
	printf("Sort of World object finish\n");
}

__global__ void make_node(Node* node, Hittable** world, size_t objectNum, u32* newOrderedIndexList)
{
	new (node) Node(world, newOrderedIndexList, 0, objectNum);
}

void RayTracingEngine::setObjects(const std::vector<Hittable*>& world)
{

	if (mRootNode)
	{
		hipFree(mRootNode);
	}

	Hittable** hittableList;

	CHECK(hipMallocManaged(&hittableList, sizeof(Hittable*) * world.size()));
	for (u32 i = 0, end = world.size(); i < end; i++)
	{
		hittableList[i] = world[i];
	}

	u32 *newOrderedIndexList;
	CHECK(hipMallocManaged(&newOrderedIndexList, sizeof(u32) * world.size()));

	sortObjects(hittableList, world.size(),newOrderedIndexList);

	CHECK(hipMalloc(&mRootNode, sizeof(Node)));
	make_node << <1, 1 >> > (mRootNode, hittableList, world.size(), newOrderedIndexList);
	CHECK(hipDeviceSynchronize());
	GPU_ERROR_CHECKER(hipPeekAtLastError());



	CHECK(hipFree(newOrderedIndexList));
}

void RayTracingEngine::setCamera(const Camera& camera)
{
	hipMemcpy(mCamera, &camera, sizeof(Camera), hipMemcpyHostToDevice);
}

void RayTracingEngine::setRenderTarget(RenderTarget& target)
{
	mRenderTarget = target;
}




__device__ Color castRayAndCalcColor(Node* worldNode, const Ray& ray, const u32 maxDepth, SecondaryInfoByRay& secondaryInfoByRay)
{
	Color resultColor(0xFFFFFF);
	Ray current_ray = ray;

	u32 hitCounter = 0;
	
	for (u32 depth = 0; depth < maxDepth; depth++)
	{
		HitRecord rec;
		if (worldNode->hit(current_ray, 0.001f, MAXFLOAT, rec))
		{
			hitCounter++;

			Ray scattered;
			Color attenuation(0x000000);
			if (rec.material->scatter(current_ray, rec, attenuation, scattered))
			{
				resultColor *= attenuation;
				current_ray = scattered;
			}
			else
			{
				secondaryInfoByRay.depth = depth;
				return attenuation;
			}
		}
		else
		{
			vec3 direction = current_ray.direction();
			f32 length2 = direction.lengthSquared();
			f32 direction_y = direction[1];
			
			f32 t = 0.5f * (tanh(direction_y * 3) + 1.0f);
			//t = tanh(t < 0 ? 0 : (t > 1 ? 1 : t));
			//printf("%f : ", t);
			resultColor *= (Color(0xFFFFFF) * t + Color(0x444444) * (1.0f - t));

			secondaryInfoByRay.depth = depth;

			return (hitCounter == 0 ? Color(0xAAAAAA) : resultColor);
		}
	}

	secondaryInfoByRay.depth = maxDepth;
	return Color(0x000000);
}


__global__ void castRayToWorld(Node* worldNode, Color* pixels, Camera* camera, const u32 screenSizeW, const u32 screenSizeH, const u32 sampleSize, const u32 maxDepth)
{
	const u32 id_w = blockIdx.x * blockDim.x + threadIdx.x;
	const u32 id_h = blockIdx.y * blockDim.y + threadIdx.y;
	if (id_h % 100 == 0 && id_w % 100 == 0) printf("%d , %d\n", id_w, id_h);
	if (id_w >= screenSizeW || id_h >= screenSizeH)
	{
		return;
	}

	const u32 pixelIndex = id_h * screenSizeW + id_w;

	const f32 inv_screenSizeW = 1.0f / static_cast<f32>(screenSizeW - 1);
	const f32 inv_screenSizeH = 1.0f / static_cast<f32>(screenSizeH - 1);

	Color resultColor = Color(0x000000);
	for (u32 s = 0; s < sampleSize; s++)
	{
		const f32 samplingRange = 0.01f;
		const f32 u = static_cast<f32>(id_w + RandomGeneratorGPU::signed_uniform_real() * samplingRange) * inv_screenSizeW;
		const f32 v = static_cast<f32>(id_h + RandomGeneratorGPU::signed_uniform_real() * samplingRange) * inv_screenSizeH;
		
		Ray ray = camera->getRay(u, v);

		SecondaryInfoByRay additinalRayInfo;
		resultColor += castRayAndCalcColor(worldNode,ray, maxDepth, additinalRayInfo);
	}
	resultColor /= sampleSize;
	
	resultColor.clamp();

	*(pixels + pixelIndex) = resultColor;
}


void RayTracingEngine::render(const u32 sampleSize, const u32 depth)
{
	printf("Rendering Start!\n");


	dim3 block(16, 16);
	dim3 grid((mRenderTarget.getResolutionWidth() + block.x - 1) / block.x, (mRenderTarget.getResolutionHeight() + block.y - 1) / block.y);
	castRayToWorld << <grid, block >> > (
		mRootNode,
		mRenderTarget.getPixels(), 
		mCamera, 
		mRenderTarget.getResolutionWidth(), 
		mRenderTarget.getResolutionHeight(),
		sampleSize,
		depth);

	CHECK(hipDeviceSynchronize());
}


void RayTracingEngine::saveRenderResult(const std::string& path)
{
	mRenderTarget.saveRenderResult(path);
}