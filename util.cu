#include <hiprand/hiprand_kernel.h>
#include "common.h"
#include "util.h"

extern __device__ hiprandState s[32];

f32 RandomGeneratorGPU::uniform_real(f32 a, f32 b)
{
	const f32 rnd = hiprand_uniform(&s[(threadIdx.x + blockIdx.x) % 32]);
	return a + (b - a) * rnd;
}

f32 RandomGeneratorGPU::signed_uniform_real(f32 a, f32 b)
{
	const f32 rnd = hiprand_uniform(&s[(threadIdx.x + blockIdx.x) % 32]);
	return a + (b - a) * rnd;
}
