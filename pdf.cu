#include "hip/hip_runtime.h"
#include "pdf.h"


f32 CosinePdf::value(const Vec3& direction) const
{
    const f32 cos0 = Vec3::dot(direction, mOnb.getAxisZ());
    return (cos0 > 0 ? cos0 / M_PI : 0);
}

Vec3 CosinePdf::generateRandomDirection() const
{
    const f32 phi = RandomGeneratorGPU::uniform_real(0, 2 * M_PI);
    const f32 u = RandomGeneratorGPU::uniform_real();

    const f32 z = sqrtf(1 - u);
    const f32 x = cos(phi) * sqrtf(u);
    const f32 y = sin(phi) * sqrtf(u);

    const Vec3 v = mOnb.local(x, y, z);

    return v;
}