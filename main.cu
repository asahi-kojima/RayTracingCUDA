#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <vector>
#include <memory>
#include <vector>
#include <string>
#include <hiprand/hiprand_kernel.h>
#include "vector.h"
#include "color.h"
#include "ray.h"
#include "object.h"
#include "util.h"
#include "hittable.h"
#include "texture.h"
#include "engine.h"

// /usr/local/cuda/bin/nvcc --generate-code arch=compute_86,code=sm_86 -std=c++17 -rdc=true -O3 -DNDEBUG -w *.cu && ./a.out && convert ./build/result.ppm ./build/rayTracingDemo.png

__device__ hiprandState s[32];

__global__ void setup_gpu()
{
	for (u32 i = 0; i < 32; i++)
	{
		hiprand_init(static_cast<unsigned long long>(i), 0, 0, &s[i]);
	}
}

int main(int argc, char** argv)
{
	hipDeviceSetLimit(hipLimitMallocHeapSize, 1024 * 1024 * 1024);
	hipDeviceSetLimit(hipLimitStackSize, 1024 * 100);
	setup_gpu<<<1, 1>>>();
	//=================================================================
	// オブジェクトの準備
	//=================================================================
	std::vector<Hittable *> world;

	constexpr s32 Range = 10;
	const vec3 center_of_all(0, 0, 0);
	for (u32 i = 0; i < 4000; i++)
	{
		const f32 max_radius = 0.3;

		const f32 theta = RandomGenerator::uniform_real() * M_PI;
		const f32 phi = RandomGenerator::uniform_real() * M_PI * 2;
		const f32 r = RandomGenerator::uniform_real() * max_radius;
		const f32 x = r * sin(theta) * cos(phi);
		const f32 y = r * sin(theta) * sin(phi);
		const f32 z = r * cos(theta);

		const vec3 center = center_of_all + vec3(x, y, z);

		const f32 extension_scale = 0.03f;

		const vec3 max_pos = vec3(RandomGenerator::uniform_real(),RandomGenerator::uniform_real(),RandomGenerator::uniform_real()) * extension_scale;
		const vec3 min_pos = vec3(RandomGenerator::uniform_real(),RandomGenerator::uniform_real(),RandomGenerator::uniform_real()) * -extension_scale;

		Material* material = make_material<Metal>(Color(RandomGenerator::uniform_int(0, 0xFFFFFF)));
		world.push_back(make_object<AABB>(center + min_pos, center + max_pos, material));
	}

	{
		vec3 center(0,100,0);
		vec3 extention = vec3(1, 0, 1) * 10000;
		world.push_back(make_object<AABB>(center - extention, center + extention,make_material<SunLight>(Color::Azure, 1)));
	}

// 	{
// 		Material* material = make_material<Dielectric>(1.5f);
// 		//material = make_material<Lambertian>(Color::Bronze);
// 		vec3 center(0,0,0);
// 		vec3 extention = vec3::one() * 1;
// #if 1
// 		world.push_back(make_object<AABB>(center - extention, center + extention,material));
// #else
// 		vec3 v0(+1, +1, +1);
// 		vec3 v1(+1, -1, +1);
// 		vec3 v2(-1, +1, +1);
// 		vec3 v3(-1, -1, +1);
// 		vec3 v4(+1, +1, -1);
// 		vec3 v5(+1, -1, -1);
// 		vec3 v6(-1, +1, -1);
// 		vec3 v7(-1, -1, -1);
// 		world.push_back(make_object<Triangle>(v3, v0, v2 , material));
// 		world.push_back(make_object<Triangle>(v3, v1, v0 , material));

// 		world.push_back(make_object<Triangle>(v1, v4, v0 , material));
// 		world.push_back(make_object<Triangle>(v1, v5, v4 , material));

// 		world.push_back(make_object<Triangle>(v7, v2, v6 , material));
// 		world.push_back(make_object<Triangle>(v7, v3, v2 , material));

// 		world.push_back(make_object<Triangle>(v5, v6, v4 , material));
// 		world.push_back(make_object<Triangle>(v5, v7, v6 , material));

// 		world.push_back(make_object<Triangle>(v2, v4, v6 , material));
// 		world.push_back(make_object<Triangle>(v2, v0, v4 , material));

// 		world.push_back(make_object<Triangle>(v7, v1, v3 , material));
// 		world.push_back(make_object<Triangle>(v7, v5, v1 , material));
// #endif
// 	}

	//=================================================================
	// カメラの準備
	//=================================================================
	constexpr f32 BaseResolution = 1.0f * 2.0f / 2;
	const u32 resolutionX = static_cast<u32>(1920 * BaseResolution);
	const u32 resolutionY = static_cast<u32>(1080 * BaseResolution);

	vec3 lookAt(0,0,0);
	// vec3 lookFrom(0.5,0.2, 1);
	// vec3 lookFrom(0.9, 0.4, 1);
	vec3 lookFrom(4.0, 1, 8);

	Camera camera = Camera(lookFrom, lookAt, vec3(0, 1, 0), 20, f32(resolutionX) / f32(resolutionY), 0.0, (lookFrom - lookAt).length());

	//=================================================================
	// レンダーターゲットの準備
	//=================================================================
	RenderTarget renderTarget[3] = {RenderTarget(resolutionX, resolutionY), RenderTarget(resolutionX, resolutionY), RenderTarget(resolutionX, resolutionY)};

	//=================================================================
	// オブジェクトの準備
	//=================================================================
	RayTracingEngine engine;

	engine.setObjects(world);
	engine.setRenderTarget(renderTarget[0]);

	camera = Camera(lookFrom, lookAt, vec3(0, 1, 0), 20, f32(resolutionX) / f32(resolutionY), 0.0, 2 * (lookFrom - lookAt).length());
	engine.setCamera(camera);
	engine.render(30, 50);

	std::string s = "./build/result";
	s += std::to_string(0);
	s += ".ppm";
	engine.saveRenderResult(s);
}
