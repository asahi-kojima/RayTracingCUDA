#include "hip/hip_runtime.h"
﻿#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include "world.h"
#include "render_target.h"
#include "engine.h"

__device__ hiprandState s[32];
__global__ void setup_gpu()
{
	for (u32 i = 0; i < 32; i++)
	{
		hiprand_init(static_cast<unsigned long long>(i), 0, 0, &s[i]);
	}
}

int main(int argc, char** argv)
{
	//hipDeviceSetLimit(hipLimitMallocHeapSize, 1024 * 1024 * 1024);
	hipDeviceSetLimit(hipLimitStackSize, 1024*100);
	setup_gpu<<<1,1>>>();
	KERNEL_ERROR_CHECKER;
	//------------------------------------------
	// レンダリングする際の解像度を外から与える
	//------------------------------------------
	if (argc <= 4)
	{
		printf("few arguments\n");
		exit(1);
	}

	const u32 ResolutionW = atoi(argv[1]);
	const u32 ResolutionH = atoi(argv[2]);
	const u32 SampleNum = atoi(argv[3]);
	const u32 MaxDepth = atoi(argv[4]);


	//------------------------------------------
	// ワールドを準備
	//------------------------------------------
	World world{};
	{
		//オブジェクトの追加
		{
			// for (s32 z = -3; z < 10; z++)
			// {
			// 	const s32 num = 30;
			// 	for (s32 i = 0; i < num * num; i++)
			// 	{
			// 		const s32 h = i / num - num/2;
			// 		const s32 w = i % num - num/2;

			// 		if (h == 0 && w == 0 && -z > 0)
			// 			continue;
			// 		f32 s = z * (M_PI) / 6;
			// 		f32 newH = cos(s) * h - sin(s) * w;
			// 		f32 newW = sin(s) * h + cos(s) * w;

			// 		Transform transform = Transform::translation(Vec3(newH, newW, -z));
			// 		transform.setRotationAngle(Vec3::generateRandomUnitVector() * 10);
			// 		transform.setScaling(0.2f);

			// 		char* primitiveName = "AABB";
			// 		char* materialName = "Metal";
			// 		if (RandomGenerator::uniform_real() < 0.3)
			// 		{
			// 			materialName = "Water";
			// 		} 

			// 		std::string objectName = "SphereObject"; objectName += std::to_string(i) += std::to_string(z);

			// 		SurfaceProperty property{};
			// 		property.setAlbedo(Color(RandomGenerator::uniform_int(0, 0xFFFFFF)));
			// 		world.addObject(objectName.c_str(), primitiveName, materialName, transform,property);
			// 	}
			// }
			
			constexpr f32 BoardScale = 555.0f;
			{
				Transform transform;
				transform.setScaling(BoardScale, 1, BoardScale);
				transform.setRotationAngle(0, 0, M_PI);
				transform.setTranslation(BoardScale / 2, BoardScale, BoardScale / 2);

				SurfaceProperty property{};
				property.setAlbedo(Color::White);

				world.addObject("Ceiling", "Board", "Lambert", transform, property);
			}

			{
				Transform transform;
				transform.setScaling(BoardScale, 1, BoardScale);
				transform.setTranslation(BoardScale / 2, 0, BoardScale / 2);

				SurfaceProperty property{};
				property.setAlbedo(Color::White);

				world.addObject("Floor", "Board", "Lambert", transform, property);
			}
			
			{
				Transform transform;
				transform.setScaling(BoardScale, 1, BoardScale);
				transform.setRotationAngle(3*M_PI_2, 0, 0);
				transform.setTranslation(BoardScale / 2, BoardScale / 2, BoardScale);

				SurfaceProperty property{};
				property.setAlbedo(Color::White);
				world.addObject("BackBoard", "Board", "Lambert", transform, property);
			}

			{
				Transform transform;
				transform.setScaling(BoardScale, 1, BoardScale);
				transform.setRotationAngle(0, 0, -M_PI_2);
				transform.setTranslation(0, BoardScale / 2, BoardScale / 2);

				SurfaceProperty property{};
				property.setAlbedo(Color::Red);
				world.addObject("RightBoard", "Board", "Lambert", transform, property);
			}

			{
				Transform transform;
				transform.setScaling(BoardScale, 1, BoardScale);
				transform.setRotationAngle(0, 0, M_PI_2);
				transform.setTranslation(BoardScale, BoardScale / 2, BoardScale / 2);

				SurfaceProperty property{};
				property.setAlbedo(Color::Green);
				world.addObject("LeftBoard", "Board", "Lambert", transform, property);
			}

			constexpr f32 BoxScale = 165.0f;
			{
				Transform transform;
				transform.setScaling(BoxScale,BoxScale, BoxScale);
				transform.setRotationAngle(0, -M_PI / 10, 0);
				const f32 angle = -M_PI / 10.0f;
				const f32 tz = cos(angle) * (BoxScale / 2) - sin(angle) * (BoxScale / 2);
				const f32 tx = sin(angle) * (BoxScale / 2) + cos(angle) * (BoxScale / 2);
				Vec3 t(tx, BoxScale / 2, tz);
				t += Vec3(130.0f, 0, 65.0f);
				transform.setTranslation(t);

				SurfaceProperty property{};
				property.setAlbedo(Color::Silver);
				world.addObject("RightBox", "AABB", "Lambert", transform, property);
			}


			{
				Transform transform;
				transform.setScaling(165.0f, 330.0, 165.0);
				transform.setRotationAngle(0, M_PI / 12, 0);
				const f32 angle = M_PI / 12.0f;
				const f32 tz = cos(angle) * (BoxScale / 2) - sin(angle) * (BoxScale / 2);
				const f32 tx = sin(angle) * (BoxScale / 2) + cos(angle) * (BoxScale / 2);
				Vec3 t(tx, 330 / 2, tz);
				t += Vec3(265.0f, 0, 295.0f);
				transform.setTranslation(t);

				SurfaceProperty property{};
				property.setAlbedo(Color::Silver);
				world.addObject("LeftBox", "AABB", "Lambert", transform, property);
			}

			{
				Transform transform;
				constexpr f32 LightSizeScale = 0.3f;
				transform.setScaling(555 * LightSizeScale, 1.0, 555 * LightSizeScale);
				transform.setRotationAngle(0, 0, -M_PI);
				Vec3 t(555 / 2, 554, 555 / 2);
				transform.setTranslation(t);

				SurfaceProperty property{};
				property.setAlbedo(Color::White);
				world.addLightObject("CeilingLight", "Board", "DiffuseLight", transform, property);
			}

			printf("Object Num in World : %d\n", world.getObjectNum());
		}

		//カメラのセット
		{
			Vec3 lookFrom(278, 278, -800);
			Vec3 lookAt(278,278,0);
			Camera camera(lookFrom, lookAt, Vec3::unitY(), 40, f32(ResolutionW) / f32(ResolutionH), 0.0, 1);
			world.setCamera(camera);
		}

		world.build();
	}


	//------------------------------------------
	// レンダーターゲットの準備
	//------------------------------------------
	RenderTarget renderTarget(ResolutionW, ResolutionH);

	//------------------------------------------
	// エンジンに渡して、レンダリング	
	//------------------------------------------
	for (u32 i = 0; i < 1; i++)
	RayTracingEngine::render(world, renderTarget, SampleNum, MaxDepth);


	//------------------------------------------
	// 画像に出力して結果の確認
	//------------------------------------------
	renderTarget.saveRenderResult("./picture/result.ppm");

}