#include "hip/hip_runtime.h"
﻿#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include "world.h"
#include "render_target.h"
#include "engine.h"

__device__ hiprandState s[32];
__global__ void setup_gpu()
{
	for (u32 i = 0; i < 32; i++)
	{
		hiprand_init(static_cast<unsigned long long>(i), 0, 0, &s[i]);
	}
}

int main(int argc, char** argv)
{
	//hipDeviceSetLimit(hipLimitMallocHeapSize, 1024 * 1024 * 1024);
	hipDeviceSetLimit(hipLimitStackSize, 1024*100);
	setup_gpu<<<1,1>>>();
	KERNEL_ERROR_CHECKER;
	//------------------------------------------
	// レンダリングする際の解像度を外から与える
	//------------------------------------------
	if (argc <= 4)
	{
		printf("few arguments\n");
		exit(1);
	}

	const u32 ResolutionW = atoi(argv[1]);
	const u32 ResolutionH = atoi(argv[2]);
	const u32 SampleNum = atoi(argv[3]);
	const u32 MaxDepth = atoi(argv[4]);


	//------------------------------------------
	// ワールドを準備
	//------------------------------------------
	World world{};
	{
		//オブジェクトの追加
		{
			for (s32 z = -3; z < 10; z++)
			{
				const s32 num = 30;
				for (s32 i = 0; i < num * num; i++)
				{
					const s32 h = i / num - num/2;
					const s32 w = i % num - num/2;

					if (h == 0 && w == 0)
						continue;
					
					Transform transform = Transform::translation(Vec3(h, w, -z));
					transform.setRotationAngle(Vec3::generateRandomUnitVector() * 10);
					transform.setScaling(0.2f);

					char* primitiveName = "AABB";
					char* materialName = "Metal";
					if (RandomGenerator::uniform_real() < 0.3)
					{
						materialName = "Diamond";
					} 

					std::string objectName = "SphereObject"; objectName += std::to_string(i) += std::to_string(z);

					SurfaceProperty property{};
					property.setAlbedo(Color(RandomGenerator::uniform_int(0, 0xFFFFFF)));
					world.addObject(objectName.c_str(), primitiveName, materialName, transform,property);
				}
			}

			printf("Object Num in World : %d\n", world.getObjectNum());
		}

		//カメラのセット
		{
			Vec3 lookAt(0, 0, 0);
			Vec3 lookFrom(0,0,5.0f);
			Camera camera(lookFrom, lookAt, Vec3::unitY(), 20, f32(ResolutionW) / f32(ResolutionH), 0.0, (lookFrom - lookAt).length());
			world.setCamera(camera);
		}

		world.buildBvh();
	}



	//------------------------------------------
	// レンダーターゲットの準備
	//------------------------------------------
	RenderTarget renderTarget(ResolutionW, ResolutionH);

	//------------------------------------------
	// エンジンに渡して、レンダリング	
	//------------------------------------------
	for (u32 i = 0; i < 1; i++)
	RayTracingEngine::render(world, renderTarget, SampleNum, MaxDepth);

	//------------------------------------------
	// 画像に出力して結果の確認
	//------------------------------------------
	renderTarget.saveRenderResult("./picture/result.ppm");

}