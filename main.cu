#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <vector>
#include <memory>
#include <vector>
#include <string>
#include <hiprand/hiprand_kernel.h>
#include "vector.h"
#include "color.h"
#include "ray.h"
#include "object.h"
#include "util.h"
#include "hittable.h"
#include "texture.h"
#include "engine.h"

// /usr/local/cuda/bin/nvcc --generate-code arch=compute_86,code=sm_86 -std=c++17 -rdc=true -O3 -DNDEBUG -w *.cu && ./a.out && convert ./build/result.ppm ./build/rayTracingDemo.png

__device__ hiprandState s[32];

__global__ void setup_gpu()
{
	for (u32 i = 0; i < 32; i++)
	{
		hiprand_init(static_cast<unsigned long long>(i), 0, 0, &s[i]);
	}
}

int main(int argc, char** argv)
{
	hipDeviceSetLimit(hipLimitMallocHeapSize, 1024 * 1024 * 1024);
	hipDeviceSetLimit(hipLimitStackSize, 1024 * 100);
	setup_gpu<<<1, 1>>>();
	//=================================================================
	// オブジェクトの準備
	//=================================================================
	World world;
	world.addObject();


	//=================================================================
	// カメラの準備
	//=================================================================
	constexpr f32 BaseResolution = 1.0f * 2.0f / 1;
	const u32 resolutionX = static_cast<u32>(1920 * BaseResolution);
	const u32 resolutionY = static_cast<u32>(1080 * BaseResolution);

	Vec3 lookAt(0,0,0);
	// Vec3 lookFrom(0.5,0.2, 1);
	// Vec3 lookFrom(0.9, 0.4, 1);
	Vec3 lookFrom(1.0, 5, 3.0);
	lookFrom *= (20 / lookFrom.length());
	Camera camera = Camera(lookFrom, lookAt, Vec3(0, 1, 0), 20, f32(resolutionX) / f32(resolutionY), 0.0, (lookFrom - lookAt).length());

	//=================================================================
	// レンダーターゲットの準備
	//=================================================================
	RenderTarget renderTarget[3] = {RenderTarget(resolutionX, resolutionY), RenderTarget(resolutionX, resolutionY), RenderTarget(resolutionX, resolutionY)};

	//=================================================================
	// オブジェクトの準備
	//=================================================================
	RayTracingEngine engine;

	engine.setObjects(world);
	engine.setRenderTarget(renderTarget[0]);

	camera = Camera(lookFrom, lookAt, Vec3(0, 1, 0), 20, f32(resolutionX) / f32(resolutionY), 0.0, 2 * (lookFrom - lookAt).length());
	engine.setCamera(camera);
	engine.render(30, 50);

	std::string s = "./build/result";
	s += std::to_string(0);
	s += ".ppm";
	engine.saveRenderResult(s);
}
