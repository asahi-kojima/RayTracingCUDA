#include "hip/hip_runtime.h"
#include "material.h"
#include "hittable.h"
#include "util.h"

//======================================================
// ランバート
//======================================================
bool Lambertian::scatter(const Ray& ray_in, const HitRecord& record, Color& attenuation, Ray& ray_scattered)
{
	const vec3 target = record.pos + record.normal + random_in_unit_sphere();
	ray_scattered.direction() = target - record.pos;
	ray_scattered.origin() = record.pos;
	attenuation = mTexture->color(0, 0, record.pos);
	return true;
}

//======================================================
// 金属
//======================================================
bool Metal::scatter(const Ray& ray_in, const HitRecord& record, Color& attenuation, Ray& ray_scattered)
{
	vec3 reflected_ray = reflect(ray_in.direction(), record.normal);

	ray_scattered = Ray(record.pos, reflected_ray + fuzz * random_in_unit_sphere() * 0.1f);

	attenuation = albedo;

	return (dot(ray_scattered.direction(), record.normal) > 0);
}


//======================================================
// 誘電体
//======================================================
bool Dielectric::scatter(const Ray& ray_in, const HitRecord& record, Color& attenuation, Ray& ray_scattered)
{
	attenuation = Color(0xFFFFFF);

	vec3 outwardNormal;
	vec3 reflected = reflect(ray_in.direction(), record.normal);
	f32 niOverNt;
	vec3 refracted;
	f32 cosine;
	f32 reflectProb;

	// 内部から出てこようとしている時
	if (dot(ray_in.direction(), record.normal) > 0)
	{
		outwardNormal = record.normal;
		niOverNt = refIdx;
		cosine = dot(ray_in.direction(), record.normal) / ray_in.direction().length();
	}
	// 外部から飛んできている時
	else
	{
		outwardNormal = -record.normal;
		niOverNt = 1.0 / refIdx;
		cosine = -dot(ray_in.direction(), record.normal) / ray_in.direction().length();
	}

	if (isRefract(ray_in.direction(), outwardNormal, niOverNt, refracted))
	{
		reflectProb = schlick(cosine, refIdx);
		if (RandomGeneratorGPU::uniform_real() < reflectProb)
		{
			ray_scattered = Ray(record.pos, reflected);
		}
		else
		{
			ray_scattered = Ray(record.pos, refracted);
		}
	}
	else
	{
		ray_scattered = Ray(record.pos, reflected);
	}

	return true;

}

bool Dielectric::isRefract(const vec3& v, const vec3& n, f32 niOverNt, vec3& refracted)
{
	vec3 uv = normalize(v);
	f32 dt = dot(uv, n);

	// スネル則を解いてる。Dはcos^2Thetaに相当し、正なら解がある。
	f32 D = 1.0 - niOverNt * niOverNt * (1 - dt * dt);

	// 解がある場合。屈折光を算出する。
	if (D > 0)
	{
		refracted = niOverNt * (uv - n * dt) + n * sqrt(D);
		return true;
	}

	// 全反射の場合
	return false;
}

f32 Dielectric::schlick(f32 cosine, f32 refIdx)
{
	f32 r0 = (1 - refIdx) / (1 + refIdx);
	r0 = r0 * r0;
	return r0 + (1 - r0) * pow((1 - cosine), 5);
}


//======================================================
// 再帰性反射素材
//======================================================
bool Retroreflective::scatter(const Ray& ray_in, const HitRecord& record, Color& attenuation, Ray& ray_scattered)
{
	ray_scattered.direction() = -ray_in.direction();
	ray_scattered.origin() = record.pos;

	attenuation = albedo;

	return false;
}


//======================================================
// 光源
//======================================================

bool SunLight::scatter(const Ray& ray_in, const HitRecord& record, Color& attenuation, Ray& ray_scattered)
{
	attenuation = Color(0xFFFFFF);
	return false;
}



//======================================================
// 重力場
//======================================================
bool GravitationalField::scatter(const Ray& ray_in, const HitRecord& record, Color& attenuation, Ray& ray_scattered)
{
	const f32 M = mGravityScale;
	const f32 m = 1.0f;
	const f32 v = 10.0f;

	const vec3 CP = record.pos - mCenter;
	const f32 R = CP.length();

	const vec3 OC = mCenter - ray_in.origin();
	const vec3 D = ray_in.direction();
	const f32 ray_center_dist_squared = (D * (dot(OC, D) / D.lengthSquared()) - OC).lengthSquared();

	const f32 E = 0.5f * m * v * v - G * M * m / R;
	const f32 L_squared = m * m * v * v * ray_center_dist_squared;
	const f32 R0 = L_squared / (G * M);
	const f32 typical_E = L_squared / (2 * R0 * R0);//典型的なエネルギースケールを意味しており、実際のエネルギーとは別

	//離心率
	const f32 e = sqrtf(1.0f + E / typical_E);
	if (e < 1.0f)
	{
		attenuation = Color(0x000000);
		return false;
	}

	attenuation = Color(0xFFFFFF);

	{
		const vec3 ux = -normalize(D);
		const vec3 uz = normalize(cross(ux, CP));
		const vec3 uy = cross(uz, ux);
		const f32 h = sqrtf(ray_center_dist_squared);
		const f32 theta = asinf(h / R);

		const f32 phi = -(acosf(((R0 / OC.length()) - 1) / e) - theta);// assert(phi < 0);
		const f32 phi2 = 2 * phi;

		const f32 x = R * cos(theta);
		const f32 y = R * sin(theta);
		const f32 cosPhi2 = cos(phi2);
		const f32 sinPhi2 = sin(phi2);

		const f32 outgoing_x = cosPhi2 * x + sinPhi2 * y;
		const f32 outgoing_y = sinPhi2 * x - cosPhi2 * y;
		const vec3 outgoing_pos = outgoing_x * ux + outgoing_y * uy + mCenter;
		const vec3 outgoing_dir = cosPhi2 * ux + sinPhi2 * uy;

		ray_scattered.direction() = outgoing_dir;
		ray_scattered.origin() = outgoing_pos;
	}
	return true;
}


//======================================================
// 疑似重力場（敢えて計算ミスを入れている）
//======================================================
bool QuasiGravitationalField::scatter(const Ray& ray_in, const HitRecord& record, Color& attenuation, Ray& ray_scattered)
{
	const f32 M = mGravityScale;
	const f32 m = 1.0f;
	const f32 v = 10.0f;

	const vec3 CP = record.pos - mCenter;
	const f32 R = CP.length();

	const vec3 OC = mCenter - ray_in.origin();
	const vec3 D = ray_in.direction();
	const f32 ray_center_dist = (D * (dot(OC, D) / D.lengthSquared()) - OC).length();

	const f32 E = 0.5f * m * v * v - G * M * m / R;
	const f32 L = m * v * ray_center_dist;
	const f32 R0 = L * L / (G * M);
	const f32 typical_E = L * L / (2 * R0 * R0);//典型的なエネルギースケールを意味しており、実際のエネルギーとは別

	//離心率
	const f32 e = sqrtf(1.0f + E / typical_E);
	if (e < 1.0f)
	{
		attenuation = Color(0x000000);
		return false;
	}

	attenuation = Color(0xFFFFFF);

	{
		const vec3 ux = -normalize(D);
		const vec3 uz = normalize(cross(ux, CP));
		const vec3 uy = cross(uz, ux);
		const f32 h = abs(dot(ux, CP));
		const f32 theta = asinf(h / R);

		const f32 phi = -(acosf(((R0 / OC.length()) - 1) / e) - theta);
		const f32 phi2 = 2 * phi;

		const f32 x = R * cos(theta);
		const f32 y = R * sin(theta);
		const f32 cosPhi2 = cos(phi2);
		const f32 sinPhi2 = sin(phi2);

		const f32 outgoing_x = cosPhi2 * x + sinPhi2 * y;
		const f32 outgoing_y = sinPhi2 * x - cosPhi2 * y;
		const vec3 outgoing_pos = outgoing_x * ux + outgoing_y * uy + mCenter;
		const vec3 outgoing_dir = cosPhi2 * ux + sinPhi2 * uy;

		ray_scattered.direction() = outgoing_dir;
		ray_scattered.origin() = outgoing_pos;
	}
	return true;
}

//======================================================
// 疑似重力場2（敢えて計算ミスを入れている）
//======================================================
bool QuasiGravitationalField2::scatter(const Ray& ray_in, const HitRecord& record, Color& attenuation, Ray& ray_scattered)
{
	const f32 M = mGravityScale;
	const f32 m = 1.0f;
	const f32 v = 10.0f;

	const vec3 CP = record.pos - mCenter;
	const f32 R = CP.length();

	const vec3 OC = mCenter - ray_in.origin();
	const vec3 D = ray_in.direction();
	const f32 ray_center_dist = (D * (dot(OC, D) / D.lengthSquared()) - OC).length();

	const f32 E = 0.5f * m * v * v - G * M * m / R;
	const f32 L = m * v * ray_center_dist;
	const f32 R0 = L * L / (G * M);
	const f32 typical_E = L * L / (2 * R0 * R0);//典型的なエネルギースケールを意味しており、実際のエネルギーとは別

	//離心率
	const f32 e = sqrtf(1.0f + E / typical_E);
	if (e < 1.0f)
	{
		attenuation = Color(0x000000);
		return false;
	}

	attenuation = Color(0xFFFFFF);

	{
		const vec3 ux = -normalize(D);
		const vec3 uz = normalize(cross(ux, CP));
		const vec3 uy = cross(uz, ux);
		const f32 h = abs(dot(uz, CP));
		const f32 theta = asinf(h / R);

		const f32 phi = -(acosf(((R0 / OC.length()) - 1) / e) - theta);
		const f32 phi2 = 2 * phi;

		const f32 x = R * cos(theta);
		const f32 y = R * sin(theta);
		const f32 cosPhi2 = cos(phi2);
		const f32 sinPhi2 = sin(phi2);

		const f32 outgoing_x = cosPhi2 * x + sinPhi2 * y;
		const f32 outgoing_y = sinPhi2 * x - cosPhi2 * y;
		const vec3 outgoing_pos = outgoing_x * ux + outgoing_y * uy + mCenter;
		const vec3 outgoing_dir = cosPhi2 * ux + sinPhi2 * uy;

		ray_scattered.direction() = outgoing_dir;
		ray_scattered.origin() = outgoing_pos;
	}
	return true;
}



//======================================================
// ラザフォード散乱
//======================================================
bool Rutherford::scatter(const Ray& ray_in, const HitRecord& record, Color& attenuation, Ray& ray_scattered)
{
	const f32 M = mGravityScale;
	const f32 m = 1.0f;
	const f32 v = 10.0f;

	const vec3 CP = record.pos - mCenter;
	const f32 R = CP.length();

	const vec3 OC = mCenter - ray_in.origin();
	const vec3 D = ray_in.direction();
	const f32 ray_center_dist = (D * (dot(OC, D) / D.lengthSquared()) - OC).length();

	const f32 E = 0.5f * m * v * v + G * M * m / R;
	const f32 L = m * v * ray_center_dist;
	const f32 R0 = L * L / (G * M);
	const f32 typical_E = L * L / (2 * R0 * R0);//典型的なエネルギースケールを意味しており、実際のエネルギーとは別

	//離心率
	const f32 e = sqrtf(1.0f + E / typical_E);

	attenuation = Color(0xFFFFFF);

	{
		const vec3 ux = -normalize(D);
		const vec3 uz = normalize(cross(ux, CP));
		const vec3 uy = cross(uz, ux);
		const f32 h = ray_center_dist;
		const f32 theta = asinf(h / R);

		const f32 phi = 2 * atan(G * M / (h * v * v));

		const f32 cosTheta = cos(theta);
		const f32 sinTheta = sin(theta);
		const f32 cosPhi = cos(phi);
		const f32 sinPhi = sin(phi);

		const f32 x = R * cosTheta;
		const f32 y = R * sinTheta;

		const f32 cosPhiTheta = cosPhi * cosTheta - sinPhi * sinTheta;//cos(phi + theta);
		const f32 sinPhiTheta = sinPhi * cosTheta + cosPhi * sinTheta;//sin(phi + theta);


		const f32 outgoing_x = -cosPhiTheta * x - sinPhiTheta * y;//-cos(phi + theta) * x + sin(phi + theta) * y;
		const f32 outgoing_y = sinPhiTheta * x - cosPhiTheta * y;//-sin(phi + theta) * x - cos(phi + theta) * y;
		const vec3 outgoing_pos = outgoing_x * ux + outgoing_y * uy + mCenter;
		const vec3 outgoing_dir = -cosPhi * ux + sinPhi * uy;

		ray_scattered.direction() = outgoing_dir;
		ray_scattered.origin() = outgoing_pos;
	}
	return true;
}

//======================================================
// 疑似ラザフォード散乱
//======================================================
bool QuasiRutherford::scatter(const Ray& ray_in, const HitRecord& record, Color& attenuation, Ray& ray_scattered)
{
	const f32 M = mGravityScale;
	const f32 m = 1.0f;
	const f32 v = 10.0f;

	const vec3 CP = record.pos - mCenter;
	const f32 R = CP.length();

	const vec3 OC = mCenter - ray_in.origin();
	const vec3 D = ray_in.direction();
	const f32 ray_center_dist = (D * (dot(OC, D) / D.lengthSquared()) - OC).length();

	const f32 E = 0.5f * m * v * v + G * M * m / R;
	const f32 L = m * v * ray_center_dist;
	const f32 R0 = L * L / (G * M);
	const f32 typical_E = L * L / (2 * R0 * R0);//典型的なエネルギースケールを意味しており、実際のエネルギーとは別

	//離心率
	const f32 e = sqrtf(1.0f + E / typical_E);

	attenuation = Color(0xFFFFFF);

	{
		const vec3 ux = -normalize(D);
		const vec3 uz = normalize(cross(ux, CP));
		const vec3 uy = cross(uz, ux);
		const f32 h = ray_center_dist;
		const f32 theta = asinf(h / R);

		const f32 phi = 2 * atan(G * M / (h * v * v));


		const f32 x = R * cos(theta);
		const f32 y = R * sin(theta);

		const f32 outgoing_x = -(cos(phi + theta) * x + sin(phi + theta) * y);
		const f32 outgoing_y = -(sin(phi + theta) * x - cos(phi + theta) * y);
		const vec3 outgoing_pos = outgoing_x * ux + outgoing_y * uy + mCenter;
		const vec3 outgoing_dir = -cos(phi) * ux + sin(phi) * uy;

		ray_scattered.direction() = outgoing_dir;
		ray_scattered.origin() = outgoing_pos;
	}
	return true;
}
